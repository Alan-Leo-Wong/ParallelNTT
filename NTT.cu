#include "hip/hip_runtime.h"
//
// Created by lei on 2023/6/2.
//
#include "NTT.hpp"
#include "hip/hip_fp16.h"
#include "cuda_util.cuh"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <random>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>

_uint128_t *rev;

void NTT::launch_cpuNTT(const _uint128_t &paddedN, _uint128_t *tempA, _uint128_t *tempB, _uint128_t *result) {
    auto cpu_NTT = [&](const bool &isInverse,
                       const _uint128_t &paddedN,
                       _uint128_t *data) {
        for (int i = 0; i < paddedN; i++)
            if (i < rev[i]) my_swap(data[i], data[rev[i]]);

        for (int i = 1; i <= L; i++) {
            _uint128_t mid = (1ULL) << (i - 1);

            _uint128_t wn = modularExponentiation(ROOT, ((MOD - 1) >> i));
            if (isInverse) wn = modularExponentiation(wn, MOD - 2);

            for (_uint128_t j = 0; j < paddedN; j += (mid << 1)) {
                _uint128_t w = 1;
                for (int k = 0; k < mid; k++, w = ((_uint128_t) w * (_uint128_t) wn) % MOD) {
                    _uint128_t x = data[j + k], y = ((_uint128_t) w * (_uint128_t) data[j + k + mid]) % MOD;
                    data[j + k] = (x + y) % MOD;
                    data[j + k + mid] = (x - y + MOD) % MOD;
                }
            }
        }
    };

    cpu_NTT(false, paddedN, tempA);
    cpu_NTT(false, paddedN, tempB);
    for (int i = 0; i < paddedN; ++i) {
        result[i] = ((_uint128_t) tempA[i] * (_uint128_t) tempB[i]) % MOD;
    }
    cpu_NTT(true, paddedN, result);
}

namespace {
    __device__ _uint128_t d_r, d_mid, d_wn;
}
__constant__ _uint128_t d_MOD = 0xFFFFFFFF00000001;
__constant__ _uint128_t d_ROOT = 7;
//__constant__ _uint128_t d_ROOT = 17492915097719143606;

__global__ void nttKernel(const _uint128_t numDivGroups, _uint128_t *d_data) {
    unsigned int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y_idx = threadIdx.y + blockIdx.y * blockDim.y;

    if (x_idx < numDivGroups && y_idx < d_mid) {
        const _uint128_t omega = modularExponentiation(d_wn, y_idx);

        _uint128_t u = d_data[x_idx * d_r + y_idx];
        _uint128_t v = d_data[x_idx * d_r + y_idx + d_mid] * omega % d_MOD;

        d_data[x_idx * d_r + y_idx] = (u + v) % d_MOD;
        d_data[x_idx * d_r + y_idx + d_mid] = (u - v + d_MOD) % d_MOD;
    }
}

__global__ void mulKernel(const _uint128_t paddedN,
                          const _uint128_t *d_tempA,
                          const _uint128_t *d_tempB,
                          _uint128_t *d_res) {
    const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < paddedN) {
        d_res[idx] = (d_tempA[idx] * d_tempB[idx]) % d_MOD;
    }
}

void NTT::launch_cuNTT(const _uint128_t &paddedN, _uint128_t *tempA, _uint128_t *tempB, _uint128_t *result) {
    auto cu_NTT = [&](const bool &isInverse,
                      const _uint128_t &paddedN,
                      _uint128_t *data) {
        for (int i = 0; i < paddedN; ++i)
            if (i < rev[i]) my_swap(data[i], data[rev[i]]);

        _uint128_t *d_data;
        CUDA_CHECK(hipMalloc((void **) &d_data, paddedN * sizeof(_uint128_t)));
        CUDA_CHECK(hipMemcpy(d_data, data, paddedN * sizeof(_uint128_t), hipMemcpyHostToDevice));

        dim3 blockSize, gridSize;
        blockSize.x = 8, blockSize.y = 128;
        for (int k = 1; k <= L; ++k) {
            _uint128_t mid = (1ULL) << (k - 1);

            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_mid), &mid, sizeof(_uint128_t)));
            _uint128_t wn = modularExponentiation(ROOT, ((MOD - 1) >> k));
            if (isInverse) wn = modularExponentiation(wn, MOD - 2);

            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_wn), &wn, sizeof(_uint128_t)));
            _uint128_t r = mid << 1;
            _uint128_t numDivGroups = (paddedN + r - 1) / r;
            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_r), &r, sizeof(_uint128_t)));

            gridSize.x = (numDivGroups + blockSize.x - 1) / blockSize.x;
            gridSize.y = (mid + blockSize.y - 1) / blockSize.y;

            nttKernel<<<gridSize, blockSize >>>(numDivGroups, d_data);
            getLastCudaError("Kernel 'nttKernel' launch failed!\n");
        }

        CUDA_CHECK(hipMemcpy(data, d_data, paddedN * sizeof(_uint128_t), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_data));
    };

    cu_NTT(false, paddedN, tempA);
    cu_NTT(false, paddedN, tempB);

    _uint128_t *d_tempA, *d_tempB, *d_res;
    CUDA_CHECK(hipMalloc((void **) &d_tempA, sizeof(_uint128_t) * paddedN));
    CUDA_CHECK(hipMemcpy(d_tempA, tempA, sizeof(_uint128_t) * paddedN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void **) &d_tempB, sizeof(_uint128_t) * paddedN));
    CUDA_CHECK(hipMemcpy(d_tempB, tempB, sizeof(_uint128_t) * paddedN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void **) &d_res, sizeof(_uint128_t) * paddedN));

    const int gridSize = (paddedN + 1024 - 1) / 1024;
    mulKernel<<<gridSize, 1024>>>(paddedN, d_tempA, d_tempB, d_res);
    CUDA_CHECK(hipMemcpy(result, d_res, sizeof(_uint128_t) * paddedN, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_tempA));
    CUDA_CHECK(hipFree(d_tempB));
    CUDA_CHECK(hipFree(d_res));

    cu_NTT(true, paddedN, result);
}

void NTT::polynomialMultiply(const TEST_TYPE &test_type,
                             const _uint128_t *coeffA,
                             const _uint128_t *coeffB,
                             TimerInterface *timer,
                             _uint128_t *result) {
    _uint128_t degreeLimit = degreeA + degreeB;
    _uint128_t paddedN = 1;
    while (paddedN <= degreeLimit) paddedN <<= 1, ++L;

    auto *tempA = new _uint128_t[paddedN];
    auto *tempB = new _uint128_t[paddedN];
    rev = new _uint128_t[paddedN];

    std::fill(tempA, tempA + paddedN, 0);
    std::fill(tempB, tempB + paddedN, 0);
    std::copy(coeffA, coeffA + degreeA + 1, tempA);
    std::copy(coeffB, coeffB + degreeB + 1, tempB);
    std::fill(rev, rev + paddedN, 0);
    for (int i = 0; i < paddedN; i++) {
        rev[i] = (rev[i >> 1] >> 1) | ((i & 1) << (L - 1));
    }

    startTimer(&timer);
    switch (test_type) {
        case CPU:
            launch_cpuNTT(paddedN, tempA, tempB, result);
            break;
        case SIMD:

            break;
        default:
            fprintf(stderr, "Unknown type! Will use CUDA.\n");
        case CUDA:
            launch_cuNTT(paddedN, tempA, tempB, result);
            break;
    }
    stopTimer(&timer);

    inv = modularExponentiation(paddedN, MOD - 2);

    delete[] tempA;
    delete[] tempB;
    delete[] rev;
}

void NTT::generateInputData(_uint128_t *coeffA, _uint128_t *coeffB) const {
    int coMin = 0, coMax = 9;
    std::random_device rd;
    std::default_random_engine engine(rd());
    std::uniform_int_distribution<int> distribution(coMin, coMax);
    // 从低到高的系数
    for (_uint128_t i = 0; i <= degreeA; ++i) {
        int x = distribution(engine);
        coeffA[i] = x;
    }
    for (_uint128_t i = 0; i <= degreeB; ++i) {
        int x = distribution(engine);
        coeffB[i] = x;
    }

    std::string filename = "input.txt";
    std::ofstream out(filename);
    if (!out) {
        fprintf(stderr, "[I/O] Line: %d Error: file %s can not be opened!\n", __LINE__, filename.c_str());
        return;
    }
    out << n << " " << m << std::endl;
    for (ull i = 0; i <= degreeA; ++i) {
        out << (int) (coeffA[i]) << " ";
    }
    out << std::endl;
    for (ull i = 0; i <= degreeB; ++i) {
        out << (int) (coeffB[i]) << " ";
    }
    out.close();
}

void NTT::run(const TEST_TYPE &type, const int &numIters) {
    TimerInterface *timer;
    createTimer(&timer);

    for (int i = 1; i <= numIters; ++i) {
        L = 0;

        auto *coeffA = new _uint128_t[degreeA + 1];
        auto *coeffB = new _uint128_t[degreeB + 1];
        generateInputData(coeffA, coeffB);

        const _uint128_t degreeLimit = degreeA + degreeB;
        auto result = new _uint128_t[degreeLimit + 1];

        polynomialMultiply(type, coeffA, coeffB, timer, result);
#ifndef NDEBUG
        std::cout << "[DEBUG] Result of Iter #" << i << ":" << std::endl;
        for (_uint128_t i = 0; i <= degreeLimit; ++i)
            std::cout << (ull) ((result[i] * inv) % MOD) << " ";
        std::cout << "\n==========\n";
#endif
        std::cout<<111<<std::endl;
        delete[] coeffA;
        delete[] coeffB;
        delete[] result;

        std::string filename = "result_" + testTypeToString(type) + ".txt";
        std::ofstream out(filename);
        if (!out) {
            fprintf(stderr, "[I/O] Line: %d Error: file %s can not be opened!\n", __LINE__, filename.c_str());
            continue;
        }
        for (_uint128_t i = 0; i <= degreeLimit; ++i)
            out << (ull) ((result[i] * inv) % MOD) << " ";
        out.close();
    }
    double avg_time = getAverageTimerValue(&timer) * 1e-3;
    printf("-- [%s] %d iterations take an average of %lf seconds\n", testTypeToString(type).c_str(), numIters,
           avg_time);

    deleteTimer(&timer);
}