#include "hip/hip_runtime.h"
//
// Created by lei on 2023/6/2.
//
#include "NTT.hpp"
#include "py_util.hpp"
#include "hip/hip_fp16.h"
#include "cuda_util.cuh"
#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <random>
#include <string>
#include <fstream>
#include <iostream>
#include <algorithm>


void NTT::launch_cpuNTT(const _uint128_t &paddedN,
                        _uint128_t *tempA,
                        _uint128_t *tempB,
                        _uint128_t *result) {
    auto cpuNtt = [&](const bool &isInverse,
                      const _uint128_t &paddedN,
                      _uint128_t *data) {
        for (int i = 0; i < paddedN; i++)
            if (i < rev[i]) my_swap(data[i], data[rev[i]]);

        for (int i = 1; i <= L; ++i) {
            _uint128_t mid = (1ULL) << (i - 1);
            _uint128_t wn = modularExponentiation(ROOT, ((MOD - 1) >> i));
            if (isInverse) wn = modularExponentiation(wn, MOD - 2);

            for (_uint128_t j = 0; j < paddedN; j += (mid << 1)) {
                _uint128_t w = 1;
                for (int k = 0; k < mid; ++k, w = (w * wn) % MOD) {
                    _uint128_t x = data[j + k], y = (w * data[j + k + mid]) % MOD;
                    data[j + k] = (x + y) % MOD;
                    data[j + k + mid] = (x - y + MOD) % MOD;
                }
            }
        }
    };

    cpuNtt(false, paddedN, tempA);
    cpuNtt(false, paddedN, tempB);
    for (int i = 0; i < paddedN; ++i) {
        result[i] = (tempA[i] * tempB[i]) % MOD;
    }
    cpuNtt(true, paddedN, result);
}

namespace {
    __device__ _uint128_t d_r, d_mid, d_wn;
}
__constant__ _uint128_t d_MOD = 0xFFFFFFFF00000001;
__constant__ _uint128_t d_ROOT = 7;
//__constant__ _uint128_t d_ROOT = 17492915097719143606;

__global__ void nttKernel(const _uint128_t numDivGroups, _uint128_t *d_data) {
    unsigned int x_idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int y_idx = threadIdx.y + blockIdx.y * blockDim.y;

    if (x_idx < d_mid && y_idx < numDivGroups) {
        const _uint128_t omega = modularExponentiation(d_wn, x_idx);

        _uint128_t u = d_data[y_idx * d_r + x_idx];
        _uint128_t v = d_data[y_idx * d_r + x_idx + d_mid] * omega % d_MOD;

        d_data[y_idx * d_r + x_idx] = (u + v) % d_MOD;
        d_data[y_idx * d_r + x_idx + d_mid] = (u - v + d_MOD) % d_MOD;
    }
}

__global__ void mulKernel(const _uint128_t paddedN,
                          const _uint128_t *d_tempA,
                          const _uint128_t *d_tempB,
                          _uint128_t *d_res) {
    const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < paddedN) {
        d_res[idx] = (d_tempA[idx] * d_tempB[idx]) % d_MOD;
    }
}

/**
 * Only to warm-up
 */
__global__ void warmUpKernel() {

}

void NTT::launch_cuNTT(const _uint128_t &paddedN,
                       _uint128_t *tempA,
                       _uint128_t *tempB,
                       _uint128_t *result) {
    auto cuNtt = [&](const bool &isInverse,
                     const _uint128_t &paddedN,
                     _uint128_t *data) {
        for (int i = 0; i < paddedN; ++i)
            if (i < rev[i]) my_swap(data[i], data[rev[i]]);

        _uint128_t *d_data;
        CUDA_CHECK(hipMalloc((void **) &d_data, paddedN * sizeof(_uint128_t)));
        CUDA_CHECK(hipMemcpy(d_data, data, paddedN * sizeof(_uint128_t), hipMemcpyHostToDevice));

        dim3 blockSize, gridSize;
        blockSize.x = 128, blockSize.y = 8;
        for (int k = 1; k <= L; ++k) {
            _uint128_t mid = (1ULL) << (k - 1);

            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_mid), &mid, sizeof(_uint128_t)));
            _uint128_t wn = modularExponentiation(ROOT, ((MOD - 1) >> k));
            if (isInverse) wn = modularExponentiation(wn, MOD - 2);

            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_wn), &wn, sizeof(_uint128_t)));
            _uint128_t r = mid << 1;
            _uint128_t numDivGroups = (paddedN + r - 1) / r;
            CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_r), &r, sizeof(_uint128_t)));

            gridSize.y = (numDivGroups + blockSize.y - 1) / blockSize.y;
            gridSize.x = (mid + blockSize.x - 1) / blockSize.x;

            nttKernel<<<gridSize, blockSize>>>(numDivGroups, d_data);
            getLastCudaError("Kernel 'nttKernel' launch failed!\n");
        }

        CUDA_CHECK(hipMemcpy(data, d_data, paddedN * sizeof(_uint128_t), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipFree(d_data));
    };

    cuNtt(false, paddedN, tempA);
    cuNtt(false, paddedN, tempB);

    _uint128_t *d_tempA, *d_tempB, *d_res;
    CUDA_CHECK(hipMalloc((void **) &d_tempA, sizeof(_uint128_t) * paddedN));
    CUDA_CHECK(hipMemcpy(d_tempA, tempA, sizeof(_uint128_t) * paddedN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void **) &d_tempB, sizeof(_uint128_t) * paddedN));
    CUDA_CHECK(hipMemcpy(d_tempB, tempB, sizeof(_uint128_t) * paddedN, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMalloc((void **) &d_res, sizeof(_uint128_t) * paddedN));

    const int gridSize = (paddedN + 1024 - 1) / 1024;
    mulKernel<<<gridSize, 1024>>>(paddedN, d_tempA, d_tempB, d_res);
    CUDA_CHECK(hipMemcpy(result, d_res, sizeof(_uint128_t) * paddedN, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(d_tempA));
    CUDA_CHECK(hipFree(d_tempB));
    CUDA_CHECK(hipFree(d_res));

    cuNtt(true, paddedN, result);
}

void NTT::polynomialMultiply(const TEST_TYPE &test_type,
                             const _uint128_t *coeffA,
                             const _uint128_t *coeffB,
                             TimerInterface *timer,
                             std::vector<_uint128_t>& result) {
    _uint128_t degreeLimit = degreeA + degreeB;
    _uint128_t paddedN = 1;
    while (paddedN <= degreeLimit) paddedN <<= 1, ++L;

    auto tempA = new _uint128_t[paddedN];
    auto tempB = new _uint128_t[paddedN];
    rev.clear(); rev.resize(paddedN, 0);
    result.resize(paddedN, 0);

    std::fill(tempA, tempA + paddedN, 0);
    std::fill(tempB, tempB + paddedN, 0);
    std::copy(coeffA, coeffA + degreeA + 1, tempA);
    std::copy(coeffB, coeffB + degreeB + 1, tempB);
//    std::fill(rev, rev + paddedN, 0);
    for (int i = 0; i < paddedN; i++) {
        rev[i] = (rev[i >> 1] >> 1) | ((i & 1) << (L - 1));
    }
    inv = modularExponentiation(paddedN, MOD - 2);

    startTimer(&timer);
    switch (test_type) {
        case CPU:
            launch_cpuNTT(paddedN, tempA, tempB, result.data());
            break;
        case SIMD:

            break;
        default:
            fprintf(stderr, "\033[1;31m[Error]\033[0m Unknown type! Will use CUDA.\n");
        case CUDA:
            launch_cuNTT(paddedN, tempA, tempB, result.data());
            break;
    }
    stopTimer(&timer);

    delete[] tempA;
    delete[] tempB;
}

void NTT::generateInputData(const std::string &in_filename,
                            _uint128_t *coeffA,
                            _uint128_t *coeffB) const {
    int coMin = 0, coMax = 9;
    std::random_device rd;
    std::default_random_engine engine(rd());
    std::uniform_int_distribution<int> distribution(coMin, coMax);

    // 从低到高的系数
    for (_uint128_t i = 0; i <= degreeA; ++i) {
        int x = distribution(engine);
        coeffA[i] = x;
    }
    for (_uint128_t i = 0; i <= degreeB; ++i) {
        int x = distribution(engine);
        coeffB[i] = x;
    }

    std::ofstream out(in_filename, std::ios::out);
    if (!out) {
        fprintf(stderr, "[I/O] Line: %d Error: file %s can not be opened!\n", __LINE__, in_filename.c_str());
        return;
    }
    out << degreeA << " " << degreeB << std::endl;
    for (ull i = 0; i <= degreeA; ++i) {
        out << (int) (coeffA[i]) << " ";
    }
    out << std::endl;
    for (ull i = 0; i <= degreeB; ++i) {
        out << (int) (coeffB[i]) << " ";
    }
    out.close();
}

void NTT::run(const TEST_TYPE &type, const int &numIters) {
    const std::string in_filename = "input.txt";
    const std::string res_filename = "result_" + testTypeToString(type) + ".txt";

    TimerInterface *timer;
    createTimer(&timer);

    if (type == TEST_TYPE::CUDA) warmUpKernel<<<1, 1>>>();
    int correct = 0;
    for (int iter = 1; iter <= numIters; ++iter) {
        L = 0;

        auto coeffA = new _uint128_t[degreeA + 1];
        auto coeffB = new _uint128_t[degreeB + 1];
//        coeffA[0] = 1, coeffA[1] = 2;
//        coeffB[0] = 1, coeffB[1] = 2, coeffB[2] = 1;
        generateInputData(in_filename, coeffA, coeffB);

        const _uint128_t degreeLimit = degreeA + degreeB;
        std::vector<_uint128_t> result;

        polynomialMultiply(type, coeffA, coeffB, timer, result);
#ifndef NDEBUG
        printf("\033[1;34m[DEBUG]\033[0m Result of Iter #%d:\n", iter);
        for (_uint128_t iter = 0; iter <= degreeLimit; ++iter)
            std::cout << (ull) ((result[iter] * inv) % MOD) << " ";
        printf("\n==========\n");
#endif
        delete[] coeffA;
        delete[] coeffB;

        std::ofstream out(res_filename, std::ios::out);
        if (!out) {
            fprintf(stderr, "[I/O] Line: %d Error: file %s can not be opened!\n", __LINE__, res_filename.c_str());
            continue;
        }
        for (_uint128_t i = 0; i <= degreeLimit; ++i)
            out << (ull) ((result[i] * inv) % MOD) << " ";
        out.close();

        try {
            std::string scriptName = R"(../eval.py)";

            // 调用Python脚本并获取返回值
            std::string py_res = runPythonScriptAndGetBoolValue(scriptName, in_filename, res_filename);

            bool boolValue = (py_res.find("True") != std::string::npos);
            if (boolValue) {
#ifndef NDEBUG
                printf("-- \033[0m\033[1;36m[INFO]\033[0m"
                       " \033[1;32m[%s]\033[0m"
                       " result at #iter %d is"
                       " \033[1;32mTRUE\033[0m\n",
                       testTypeToString(type).c_str(), iter);
#endif
                ++correct;
            } else {
#ifndef NDEBUG
                printf("-- \033[0m\033[1;36m[INFO]\033[0m"
                       " \033[1;32m[%s]\033[0m"
                       " result at #iter %d is"
                       " \033[1;31mFALSE\033[0m\n",
                       testTypeToString(type).c_str(), iter);
#endif
            }
        } catch (const std::exception &e) {
            std::cerr << "-- \033[0m\033[1;31m[Error]\033[0m " << e.what() << std::endl;
        }
    }
    double avg_time = getAverageTimerValue(&timer) * 1e-3;
    printf("-- \033[0m\033[1;36m[INFO]\033[0m"
           " \033[0m\033[1;32m[%s]\033[0m"
           " %d iterations take an average of"
           " \033[1;31m%lf\033[0m"
           " seconds,"
           " correct rate ="
           " \033[1;31m%.2lf%%\033[0m\n",
           testTypeToString(type).c_str(),
           numIters, avg_time, correct * 100.0 / numIters);

    deleteTimer(&timer);
}